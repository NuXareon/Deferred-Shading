#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include "glm.hpp"

#define BLOCK_SIZE				256
int adsds = 3;

int *__d_res, *__d_nLights, *__d_lightsScanSum, *__d_lightsMatrix, *__d_gridRes, *__d_lightsTile, *__d_gLightsCol, *__d_lightsMatrixCompact, *__d_lightsScanSumLength;
float *__d_pla, *__d_threshold;
glm::mat4 *__d_m, *__d_proj;
glm::vec3 *__d_lightsProj, *__d_r;

__device__ float calcLightRadius(float* l, float threshold)
{
	if (l[11] == 0.0f) return (threshold*l[3]*glm::max(glm::max(l[0],l[1]),l[2])-l[9])/l[10];
	else return (sqrt(-4*l[9]*l[11]+l[10]*l[10]+4*l[11]*threshold*l[3]*glm::max(glm::max(l[0],l[1]),l[2]))-l[10])/(2*l[11]);
}


__global__ void calcLightProj(float *d_pla, float* d_threshold, glm::vec3* d_r, glm::mat4* d_m, glm::mat4* d_proj, glm::vec3* d_lightsProj, int* d_res, int* nLights)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int n = *nLights;

	if (idx < n) {
		glm::vec4 c = glm::vec4(d_pla[idx*12+6], d_pla[idx*12+7], d_pla[idx*12+8],1.0);
		float r = calcLightRadius(&d_pla[idx*12], *d_threshold);
	
		glm::vec4 p = c+r*glm::vec4(*d_r,0.0);
		p.w=1.0f;

		glm::mat4 m = *d_m;
		glm::mat4 proj = *d_proj;
	
		glm::vec4 cp = proj*m*c;
		cp = cp/cp.w;
		cp.x = d_res[0]*(cp.x+1)/2;
		cp.y = d_res[1]*(cp.y+1)/2;
	
		glm::vec4 pp = proj*m*p;
		pp = pp/pp.w;
		pp.x = d_res[0]*(pp.x+1)/2;
		pp.y = d_res[1]*(pp.y+1)/2;
	
		float pRadius = glm::length(glm::vec2(cp.x,cp.y)-glm::vec2(pp.x,pp.y));

		d_lightsProj[idx] = glm::vec3(cp.x,cp.y,pRadius);
	}
}

__global__ void calcLightsMatrix(glm::vec3* lightsProj, int* lightsScanSum, int* lightsMatrix, int* _nLights, int* _gridRes, int* _lightsTile, int* _gLightsCol)
{
	int gridRes = *_gridRes;
	int lightsTile = *_lightsTile;
	int gLightsCol = *_gLightsCol;
	int nLights = *_nLights;

	int j = blockIdx.x;
	int k = blockIdx.y;
	int x1 = k*gridRes; int x2 = (k+1)*gridRes;
	int y1 = j*gridRes; int y2 = (j+1)*gridRes;
	int off;
	for (int i = 0; i < nLights ; ++i)
	{
		glm::vec2 cp = glm::vec2(lightsProj[i].x,lightsProj[i].y);
		float pRadius = lightsProj[i].z;
		if (cp.x >= x1 && cp.x < x2) {
			if (cp.y >= y1 && cp.y < y2) { //center inside tile
				off = lightsScanSum[j*gLightsCol+k]++; // comprovacio que no ens pasem de max num ligths per tile
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			} 
			else if (cp.y > y2) { // down (r> y-y2)
				if (pRadius >= abs(cp.y-y2)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			} 
			else if (cp.y < y1) { // up (r> y-y1)
				if (pRadius >= abs(cp.y-y1)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			} 
		} 
		else if (cp.y >= y1 && cp.y < y2) { 
			if (cp.x < x1) { // left
				if (pRadius >= abs(cp.x-x1)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			}
			else if (cp.x > x2) { // right
				if (pRadius >= abs(cp.x-x2)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			}
		}
		else if (cp.x < x1 && cp.y < y1) { // upper-left
			if (pRadius >= sqrt((cp.x-x1)*(cp.x-x1)+(cp.y-y1)*(cp.y-y1))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		} 
		else if (cp.x > x2 && cp.y < y1) { // upper-right
			if (pRadius >= sqrt((cp.x-x2)*(cp.x-x2)+(cp.y-y1)*(cp.y-y1))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		}
		else if (cp.x < x1 && cp.y > y2) { // down-left
			if (pRadius >= sqrt((cp.x-x1)*(cp.x-x1)+(cp.y-y2)*(cp.y-y2))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		}
		else if (cp.x > x2 && cp.y > y2) { // down-right
			if (pRadius >= sqrt((cp.x-x2)*(cp.x-x2)+(cp.y-y2)*(cp.y-y2))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		}
	}
}

__global__ void compactLightsMatrix(int* lightsScanSum, int* lightsMatrix, int* lightsMatrixCompact, int* _lightsTile, int* _lightsScanSumLength)
{
	int lightsScanSumLength = *_lightsScanSumLength;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < lightsScanSumLength) {
		int lightsTile = *_lightsTile;
		int prev;
		if (idx == 0) prev = 0;
		else prev = lightsScanSum[idx-1];
		int act = lightsScanSum[idx];
		for (int i = 0; i < act-prev; ++i) {
			lightsMatrixCompact[prev+i] = lightsMatrix[idx*lightsTile+i];
		}
	}
	/*
	int* partialLightsMatrix;
	partialLightsMatrix = new int[act-prev];

	for (int i = 0; i < act-prev; ++i) {
		partialLightsMatrix[i] = lightsMatrix[idx*lightsTile+i];
	}
	
	__syncthreads();
	for (int i = 0; i < act-prev; ++i) {
		lightsMatrix[prev+i] = partialLightsMatrix[i];
	}
	*/
}

extern "C" void initMemCUDA(int **d_res, int **d_nLights, int **d_lightsScanSum, int **d_lightsMatrix, int **d_gridRes, int **d_lightsTile, int **d_gLightsCol, int **d_lightsMatrixCompact, int **d_lightsScanSumLength,
	float **d_pla, float **d_threshold,
	glm::mat4 **d_m, glm::mat4 **d_proj,
	glm::vec3 **d_lightsProj, glm::vec3 **d_r,
	int nLights, int lightsScanSumLength, int lightsMatrixLength)
{
	hipError_t err;
	err=hipMalloc((void **) d_pla, nLights*12*sizeof(float)); //[r,g,b, i,i_,i__, x,y,z, con,lin,exp]
	err=hipMalloc((void **) d_threshold, sizeof(float));
	hipMalloc((void **) d_r, sizeof(glm::vec3));
	hipMalloc((void **) d_m, sizeof(glm::mat4));
	hipMalloc((void **) d_proj, sizeof(glm::mat4));
	err=hipMalloc((void **) d_lightsProj, nLights*sizeof(glm::vec3)); // [x,y,radius]
	err=hipMalloc((void **) d_res, 2*sizeof(int));
	err=hipMalloc((void **) d_nLights, sizeof(int));
	hipMalloc((void **) d_gridRes, sizeof(int));
	hipMalloc((void **) d_lightsTile, sizeof(int));
	hipMalloc((void **) d_gLightsCol, sizeof(int));
	hipMalloc((void **) d_lightsScanSumLength, sizeof(int));
	hipMalloc((void **) d_lightsScanSum, lightsScanSumLength*sizeof(int));
	hipMalloc((void **) d_lightsMatrix, lightsMatrixLength*sizeof(int));
	hipMalloc((void **) d_lightsMatrixCompact, lightsMatrixLength*sizeof(int));

	err=hipMalloc((void **) &__d_pla, nLights*12*sizeof(float)); //[r,g,b, i,i_,i__, x,y,z, con,lin,exp]
	err=hipMalloc((void **) &__d_threshold, sizeof(float));
	err=hipMalloc((void **) &__d_r, sizeof(glm::vec3));
	err=hipMalloc((void **) &__d_m, sizeof(glm::mat4));
	err=hipMalloc((void **) &__d_proj, sizeof(glm::mat4));
	err=hipMalloc((void **) &__d_lightsProj, nLights*sizeof(glm::vec3)); // [x,y,radius]
	err=hipMalloc((void **) &__d_res, 2*sizeof(int));
	err=hipMalloc((void **) &__d_nLights, sizeof(int));
	err=hipMalloc((void **) &__d_gridRes, sizeof(int));
	err=hipMalloc((void **) &__d_lightsTile, sizeof(int));
	err=hipMalloc((void **) &__d_gLightsCol, sizeof(int));
	err=hipMalloc((void **) &__d_lightsScanSumLength, sizeof(int));
	err=hipMalloc((void **) &__d_lightsScanSum, lightsScanSumLength*sizeof(int));
	err=hipMalloc((void **) &__d_lightsMatrix, lightsMatrixLength*sizeof(int));
	err=hipMalloc((void **) &__d_lightsMatrixCompact, lightsMatrixLength*sizeof(int));
}

extern "C" void freeMemCUDA(int *d_res, int *d_nLights, int *d_lightsScanSum, int *d_lightsMatrix, int *d_gridRes, int *d_lightsTile, int *d_gLightsCol, int *d_lightsMatrixCompact, int *d_lightsScanSumLength,
	float *d_pla, float *d_threshold,
	glm::mat4 *d_m, glm::mat4 *d_proj,
	glm::vec3 *d_lightsProj, glm::vec3 *d_r)
{
	hipFree(d_pla);
	hipFree(d_threshold);
	hipFree(d_r);
	hipFree(d_m);
	hipFree(d_proj);
	hipFree(d_res);
	hipFree(d_lightsProj);
	hipFree(d_nLights);
	hipFree(d_gridRes);
	hipFree(d_gLightsCol);
	hipFree(d_lightsScanSumLength);
	hipFree(d_lightsMatrixCompact);
	hipFree(d_lightsTile);
	hipFree(d_lightsScanSum);
	hipFree(d_lightsMatrix);
}

extern "C" void launch_kernel(void* pointLightsArr, int nLights, float threshold, glm::vec3* right, float* gl_ModelViewMatrix, float* gl_ProjectionMatrix, int w, int h,
	int gLightsRow, int gLightsCol, int* lightsScanSum, int lightsScanSumLength, int* lightsMatrix, int lightsMatrixLength, int gridRes, int lightsTile, int *d_lightsScanSumLength,
	int *d_res, int *d_nLights, int *d_lightsScanSum, int *d_lightsMatrix, int *d_gridRes, int *d_lightsTile, int *d_gLightsCol, int *d_lightsMatrixCompact,
	float *d_pla, float *d_threshold,
	glm::mat4 *d_m, glm::mat4 *d_proj,
	glm::vec3 *d_lightsProj, glm::vec3 *d_r)
{
	
	int *_d_res, *_d_nLights, *_d_lightsScanSum, *_d_lightsMatrix, *_d_gridRes, *_d_lightsTile, *_d_gLightsCol, *_d_lightsMatrixCompact, *_d_lightsScanSumLength;
	float *_d_pla, *_d_threshold;
	glm::mat4 *_d_m, *_d_proj;
	glm::vec3 *_d_lightsProj, *_d_r;
	
	/*
	float plaTest[100];
	float* plaFloat = (float*)pointLightsArr; 
	int ssTest[100];
	for (int i = 0; i < 100; ++i) {
		plaTest[i] = plaFloat[i];
		ssTest[i] = lightsScanSum[i];
	}
	float mv[16],pm[16];
	for (int i = 0; i < 16; ++i) {
		mv[i] = gl_ModelViewMatrix[i];
		pm[i] = gl_ProjectionMatrix[i];
	}
	*/
	hipError_t err;
//	float *_d_pla;
 
	
	/*
	int *d_res, *d_nLights, *d_lightsScanSum, *d_lightsMatrix, *d_gridRes, *d_lightsTile, *d_gLightsCol, *d_lightsMatrixCompact;
	float *d_pla,*d_threshold;
	glm::mat4 *d_m,*d_proj;
	glm::vec3 *d_lightsProj,*d_r;
	*/
	// Calc lights projection
	glm::mat4 m = glm::mat4(gl_ModelViewMatrix[0],gl_ModelViewMatrix[1],gl_ModelViewMatrix[2],gl_ModelViewMatrix[3],gl_ModelViewMatrix[4],gl_ModelViewMatrix[5],gl_ModelViewMatrix[6],gl_ModelViewMatrix[7],gl_ModelViewMatrix[8],gl_ModelViewMatrix[9],gl_ModelViewMatrix[10],gl_ModelViewMatrix[11],gl_ModelViewMatrix[12],gl_ModelViewMatrix[13],gl_ModelViewMatrix[14],gl_ModelViewMatrix[15]);
	glm::mat4 proj = glm::mat4(gl_ProjectionMatrix[0],gl_ProjectionMatrix[1],gl_ProjectionMatrix[2],gl_ProjectionMatrix[3],gl_ProjectionMatrix[4],gl_ProjectionMatrix[5],gl_ProjectionMatrix[6],gl_ProjectionMatrix[7],gl_ProjectionMatrix[8],gl_ProjectionMatrix[9],gl_ProjectionMatrix[10],gl_ProjectionMatrix[11],gl_ProjectionMatrix[12],gl_ProjectionMatrix[13],gl_ProjectionMatrix[14],gl_ProjectionMatrix[15]);
	int h_res[2] = {w,h};
	int bdsf = adsds;
	/*
	float *d_pla2;
	int *d_res2;
	err=hipMalloc((void**) &d_pla2, nLights*12*sizeof(float));
	err=hipMalloc((void**) &d_res2, sizeof(h_res));
	err=hipMemcpy(d_pla2,pointLightsArr, nLights*12*sizeof(float),hipMemcpyHostToDevice);
	err=hipMemcpy(d_res2,h_res, sizeof(h_res),hipMemcpyHostToDevice);
	*/
	/*
	hipMalloc((void **) &d_pla, nLights*12*sizeof(float)); //[r,g,b, i,i_,i__, x,y,z, con,lin,exp]
	hipMalloc((void **) &d_threshold, sizeof(float));
	hipMalloc((void **) &d_r, sizeof(glm::vec3));
	hipMalloc((void **) &d_m, sizeof(glm::mat4));
	hipMalloc((void **) &d_proj, sizeof(glm::mat4));
	hipMalloc((void **) &d_lightsProj, nLights*sizeof(glm::vec3)); // [x,y,radius]
	hipMalloc((void **) &d_res, sizeof(h_res));
	*/

	err=hipMalloc((void **) &_d_pla, nLights*12*sizeof(float)); //[r,g,b, i,i_,i__, x,y,z, con,lin,exp]
	err=hipMalloc((void **) &_d_threshold, sizeof(float));
	hipMalloc((void **) &_d_r, sizeof(glm::vec3));
	hipMalloc((void **) &_d_m, sizeof(glm::mat4));
	hipMalloc((void **) &_d_proj, sizeof(glm::mat4));
	err=hipMalloc((void **) &_d_lightsProj, nLights*sizeof(glm::vec3)); // [x,y,radius]
	err=hipMalloc((void **) &_d_res, 2*sizeof(int));
	err=hipMalloc((void **) &_d_nLights, sizeof(int));
	hipMalloc((void **) &_d_gridRes, sizeof(int));
	hipMalloc((void **) &_d_lightsTile, sizeof(int));
	hipMalloc((void **) &_d_gLightsCol, sizeof(int));
	hipMalloc((void **) &_d_lightsScanSumLength, sizeof(int));
	hipMalloc((void **) &_d_lightsScanSum, lightsScanSumLength*sizeof(int));
	hipMalloc((void **) &_d_lightsMatrix, lightsMatrixLength*sizeof(int));
	hipMalloc((void **) &_d_lightsMatrixCompact, lightsMatrixLength*sizeof(int));

	err=hipMemcpy(_d_pla,pointLightsArr, nLights*12*sizeof(float),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_threshold,&threshold, sizeof(float),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_r,right, sizeof(glm::vec3),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_m,&m, sizeof(glm::mat4),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_proj,&proj, sizeof(glm::mat4),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_res,&h_res, sizeof(h_res),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_nLights,&nLights, sizeof(int),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_gridRes,&gridRes, sizeof(int),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_lightsTile,&lightsTile, sizeof(int),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_gLightsCol,&gLightsCol, sizeof(int),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_lightsScanSum,lightsScanSum, lightsScanSumLength*sizeof(int),hipMemcpyHostToDevice);
	err=hipMemcpy(_d_lightsScanSumLength,&lightsScanSumLength, sizeof(int),hipMemcpyHostToDevice);
	/*
	int sizeVec3 = sizeof(glm::vec3);
	int sizeMat4 = sizeof(glm::mat4);
	int sizeHres = sizeof(h_res);
	int *lightsScanSumHost;
	lightsScanSumHost = new int[lightsScanSumLength];
	int lsstest[100];
	glm::vec3 projTest[20];
	float testthresh = 0;
	glm::vec3 rightTest;
	int testnl = 0;
	int lttest = 0;
	int lctest = 0;
	int bla[2];
	
	err=hipMemcpy(plaTest,d_pla, 100*sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testthresh,d_threshold, sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&rightTest,d_r, sizeof(glm::vec3),hipMemcpyDeviceToHost);
	err=hipMemcpy(&m,d_m, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&proj,d_proj, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&h_res,d_res, sizeof(h_res),hipMemcpyDeviceToHost);

	err=hipMemcpy(projTest,d_lightsProj, 20*sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(ssTest,d_lightsScanSum, 100*sizeof(int),hipMemcpyDeviceToHost);
	//err=hipMemcpy(lightsMatrix,_d_lightsMatrixCompact, lightsMatrixLength*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testnl,d_nLights, sizeof(int),hipMemcpyDeviceToHost);
	//err=hipMemcpy(bla,_d_gridRes, 2*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lttest ,d_lightsTile, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lctest ,d_gLightsCol, sizeof(int),hipMemcpyDeviceToHost);
	*/
	int nBlocks = glm::ceil((float)nLights/BLOCK_SIZE);
	
	calcLightProj<<<nBlocks,BLOCK_SIZE>>>(_d_pla, _d_threshold, _d_r, _d_m, _d_proj, _d_lightsProj, _d_res, _d_nLights); //O(1)
	/*
	err=hipMemcpy(plaTest,_d_pla, 100*sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testthresh,_d_threshold, sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&rightTest,_d_r, sizeof(glm::vec3),hipMemcpyDeviceToHost);
	err=hipMemcpy(&m,_d_m, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&proj,_d_proj, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&h_res,_d_res, sizeof(h_res),hipMemcpyDeviceToHost);

	err=hipMemcpy(projTest,_d_lightsProj, 20*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	err=hipMemcpy(ssTest,_d_lightsScanSum, 100*sizeof(int),hipMemcpyDeviceToHost);
	//err=hipMemcpy(lightsMatrix,_d_lightsMatrixCompact, lightsMatrixLength*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testnl,_d_nLights, sizeof(int),hipMemcpyDeviceToHost);
	//err=hipMemcpy(bla,_d_gridRes, 2*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lttest ,_d_lightsTile, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lctest ,_d_gLightsCol, sizeof(int),hipMemcpyDeviceToHost);
	*/
	/*
	err=hipMemcpy(projTest,_d_lightsProj, nLights*3,hipMemcpyDeviceToHost);
	err=hipMemcpy(ssTest,_d_lightsScanSum, lightsScanSumLength*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(lightsMatrix,_d_lightsMatrixCompact, lightsMatrixLength*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testnl,_d_nLights, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(bla,_d_gridRes, 2*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lttest ,_d_lightsTile, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lctest ,_d_gLightsCol, sizeof(int),hipMemcpyDeviceToHost);
	*/
	/*
	hipFree(d_pla);
	hipFree(d_threshold);
	hipFree(d_r);
	hipFree(d_m);
	hipFree(d_proj);
	hipFree(d_res);
	
	// Calc lights matrix
	hipMalloc((void **) &d_nLights, sizeof(int));
	hipMalloc((void **) &d_gridRes, sizeof(int));
	hipMalloc((void **) &d_lightsTile, sizeof(int));
	hipMalloc((void **) &d_gLightsCol, sizeof(int));
	hipMalloc((void **) &d_lightsScanSum, lightsScanSumLength*sizeof(int));
	hipMalloc((void **) &d_lightsMatrix, lightsMatrixLength*sizeof(int));
	*/
	/*
	int gridResTest;
	int lMatrixTest[100];
	size_t free_byte ;
    size_t total_byte ;
	hipMemGetInfo( &free_byte, &total_byte );
	*/
	calcLightsMatrix<<<dim3(gLightsRow, gLightsCol),1>>>(_d_lightsProj, _d_lightsScanSum, _d_lightsMatrix, _d_nLights, _d_gridRes, _d_lightsTile, _d_gLightsCol); //O(n)
	
	/*
	hipMemGetInfo( &free_byte, &total_byte );
	
	err=hipMemcpy(plaTest,_d_pla, 100*sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testthresh,_d_threshold, sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&rightTest,_d_r, sizeof(glm::vec3),hipMemcpyDeviceToHost);
	err=hipMemcpy(&m,_d_m, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&proj,_d_proj, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&h_res,_d_res, sizeof(h_res),hipMemcpyDeviceToHost);

	err=hipMemcpy(projTest,_d_lightsProj, 20*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	err=hipMemcpy(ssTest,_d_lightsScanSum, 100*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(lMatrixTest,_d_lightsMatrix, 100*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testnl,_d_nLights, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&gridResTest,_d_gridRes, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lttest ,_d_lightsTile, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lctest ,_d_gLightsCol, sizeof(int),hipMemcpyDeviceToHost);

	int asdfasdf = 0;
	
	/*
	hipFree(d_lightsProj);
	hipFree(d_nLights);
	hipFree(d_gridRes);
	hipFree(d_gLightsCol);
	*/
	// Scan and compact
	thrust::inclusive_scan(thrust::device, _d_lightsScanSum, _d_lightsScanSum + lightsScanSumLength, _d_lightsScanSum);
	//rr=hipMemcpy(ssTest,d_lightsScanSum, 100*sizeof(int),hipMemcpyDeviceToHost);
	/*
	int ligtshMatrixCompactSize = thrust::reduce(lightsScanSum,lightsScanSum+lightsScanSumLength,-1,thrust::maximum<int>());

	hipMalloc((void **) &d_lightsMatrixCompact, ligtshMatrixCompactSize*sizeof(int));
	*/
	
	nBlocks = glm::ceil((float)lightsScanSumLength/BLOCK_SIZE);

	compactLightsMatrix<<<nBlocks,BLOCK_SIZE>>>(_d_lightsScanSum, _d_lightsMatrix, _d_lightsMatrixCompact, _d_lightsTile, _d_lightsScanSumLength); //O(1)
	/*
	int lmCompact[100];
	
	err=hipMemcpy(plaTest,_d_pla, 100*sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testthresh,_d_threshold, sizeof(float),hipMemcpyDeviceToHost);
	err=hipMemcpy(&rightTest,_d_r, sizeof(glm::vec3),hipMemcpyDeviceToHost);
	err=hipMemcpy(&m,_d_m, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&proj,_d_proj, sizeof(glm::mat4),hipMemcpyDeviceToHost);
	err=hipMemcpy(&h_res,_d_res, sizeof(h_res),hipMemcpyDeviceToHost);

	err=hipMemcpy(projTest,_d_lightsProj, 20*sizeof(glm::vec3),hipMemcpyDeviceToHost);
	err=hipMemcpy(ssTest,_d_lightsScanSum, 100*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(lMatrixTest,_d_lightsMatrix, 100*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(lmCompact,_d_lightsMatrixCompact, 100*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&testnl,_d_nLights, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&gridResTest,_d_gridRes, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lttest ,_d_lightsTile, sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(&lctest ,_d_gLightsCol, sizeof(int),hipMemcpyDeviceToHost);
	*/

	err=hipMemcpy(lightsScanSum ,_d_lightsScanSum, lightsScanSumLength*sizeof(int),hipMemcpyDeviceToHost);
	err=hipMemcpy(lightsMatrix ,_d_lightsMatrixCompact, lightsMatrixLength*sizeof(int),hipMemcpyDeviceToHost);
	
	/*
	int a = lightsScanSum[3];
	int b = lightsScanSum[4];
	int c = lightsMatrix[3];
	int d = lightsMatrix[4];

	
	hipFree(d_lightsMatrixCompact);
	hipFree(d_lightsTile);
	hipFree(d_lightsScanSum);
	hipFree(d_lightsMatrix);
	*/

	//hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte );

	hipFree(_d_pla);
	hipFree(_d_threshold);
	hipFree(_d_r);
	hipFree(_d_m);
	hipFree(_d_proj);
	hipFree(_d_res);
	hipFree(_d_lightsProj);
	hipFree(_d_nLights);
	hipFree(_d_gridRes);
	hipFree(_d_gLightsCol);
	hipFree(_d_lightsScanSumLength);
	hipFree(_d_lightsMatrixCompact);
	hipFree(_d_lightsTile);
	hipFree(_d_lightsScanSum);
	hipFree(_d_lightsMatrix);
	int asd=2;
}