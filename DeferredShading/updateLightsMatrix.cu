#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include "glm.hpp"

#define BLOCK_SIZE				256

int *d_lightsMatrix, *d_lightsScanSum, *d_lightsMatrixCompact;
float *d_pla;
glm::vec3 *d_lightsProj;
hipError_t err;
size_t size;

__device__ float calcLightRadius(float* l, float threshold)
{
	if (l[11] == 0.0f) return (threshold*l[3]*glm::max(glm::max(l[0],l[1]),l[2])-l[9])/l[10];
	else return (sqrt(-4*l[9]*l[11]+l[10]*l[10]+4*l[11]*threshold*l[3]*glm::max(glm::max(l[0],l[1]),l[2]))-l[10])/(2*l[11]);
}


__global__ void calcLightProj(float *d_pla, float threshold, glm::vec3 right, glm::mat4 m, glm::mat4 proj, glm::vec3* d_lightsProj, int w, int h, int nLights)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx < nLights) {
		glm::vec4 c = glm::vec4(d_pla[idx*12+6], d_pla[idx*12+7], d_pla[idx*12+8],1.0);
		float r = calcLightRadius(&d_pla[idx*12], threshold);
	
		glm::vec4 p = c+r*glm::vec4(right,0.0);
		p.w=1.0f;

		glm::vec4 cp = proj*m*c;
		cp = cp/cp.w;
		cp.x = w*(cp.x+1)/2;
		cp.y = h*(cp.y+1)/2;
	
		glm::vec4 pp = proj*m*p;
		pp = pp/pp.w;
		pp.x = w*(pp.x+1)/2;
		pp.y = h*(pp.y+1)/2;
	
		float pRadius = glm::length(glm::vec2(cp.x,cp.y)-glm::vec2(pp.x,pp.y));

		d_lightsProj[idx] = glm::vec3(cp.x,cp.y,pRadius);
	}
}

__global__ void calcLightsMatrix(glm::vec3* lightsProj, int* lightsMatrix, int nLights, int gridRes, int lightsTile, int gLightsRow, int gLightsCol)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx < nLights){
		glm::vec2 cp = glm::vec2(lightsProj[idx].x,lightsProj[idx].y);
		float pRadius = lightsProj[idx].z;

		float x1,x2,y1,y2;
		for (unsigned int j = 0; j < gLightsRow; j++) {
			for (unsigned int k = 0; k < gLightsCol; k++) {
				x1 = k*gridRes; x2 = (k+1)*gridRes;
				y1 = j*gridRes; y2 = (j+1)*gridRes;

				lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 0;
				if (cp.x >= x1 && cp.x < x2) {
					if (cp.y >= y1 && cp.y < y2) { //center inside tile
						lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
					} 
					else if (cp.y > y2) { // down (r> y-y2)
						if (pRadius >= abs(cp.y-y2)) {
							lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
						}
					} 
					else if (cp.y < y1) { // up (r> y-y1)
						if (pRadius >= abs(cp.y-y1)) {
							lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
						}
					} 
				} 
				else if (cp.y >= y1 && cp.y < y2) { 
					if (cp.x < x1) { // left
						if (pRadius >= abs(cp.x-x1)) {
							lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
						}
					}
					else if (cp.x > x2) { // right
						if (pRadius >= abs(cp.x-x2)) {
							lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
						}
					}
				}
				else if (cp.x < x1 && cp.y < y1) { // upper-left
					if (pRadius >= sqrt((cp.x-x1)*(cp.x-x1)+(cp.y-y1)*(cp.y-y1))) {
						lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
					}
				} 
				else if (cp.x > x2 && cp.y < y1) { // upper-right
					if (pRadius >= sqrt((cp.x-x2)*(cp.x-x2)+(cp.y-y1)*(cp.y-y1))) {
						lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
					}
				}
				else if (cp.x < x1 && cp.y > y2) { // down-left
					if (pRadius >= sqrt((cp.x-x1)*(cp.x-x1)+(cp.y-y2)*(cp.y-y2))) {
						lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
					}
				}
				else if (cp.x > x2 && cp.y > y2) { // down-right
					if (pRadius >= sqrt((cp.x-x2)*(cp.x-x2)+(cp.y-y2)*(cp.y-y2))) {
						lightsMatrix[(j*gLightsCol+k)*lightsTile+idx] = 1;
					}
				}
			}
		}
	}
}

__global__ void preScanSum(int *lightsScanSum, int *lightsMatrix, int lightsScanSumLength, int lightsTile, int nLights)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx < lightsScanSumLength) {
		for (int i = 0; i < nLights; ++i) {
			if (lightsMatrix[idx*lightsTile+i] == 1) lightsScanSum[idx]++;
		}
	}
}

__global__ void compactLightsMatrix(int* lightsScanSum, int* lightsMatrix, int* lightsMatrixCompact, int lightsTile, int lightsScanSumLength, int nLights)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < lightsScanSumLength) {
		int prev;
		if (idx == 0) prev = 0;
		else prev = lightsScanSum[idx-1];
		int act = lightsScanSum[idx];
		int off = 0;
		for (int i = 0; i < nLights; ++i) {
			if (lightsMatrix[idx*lightsTile+i] == 1) {
				lightsMatrixCompact[prev+off] = i;
				off++;
			}
		}
	}
}

extern "C" void initMemCUDA(int nLights, int lightsScanSumLength, int lightsMatrixLength)
{
	hipMalloc((void **) &d_pla, nLights*12*sizeof(float)); //[r,g,b, i,i_,i__, x,y,z, con,lin,exp]
	hipMalloc((void **) &d_lightsProj, nLights*sizeof(glm::vec3)); // [x,y,radius]
	hipMalloc((void **) &d_lightsMatrix, lightsMatrixLength*sizeof(int));
}

extern "C" void freeMemCUDA()
{
	hipFree(d_pla);;
	hipFree(d_lightsProj);
	hipFree(d_lightsMatrix);
}

extern "C" void launch_kernel(void* pointLightsArr, int nLights, float threshold, glm::vec3 right, float* gl_ModelViewMatrix, float* gl_ProjectionMatrix, int w, int h,
	int gLightsRow, int gLightsCol, int* lightsScanSum, int lightsScanSumLength, int* lightsMatrix, int lightsMatrixLength, int gridRes, int lightsTile, hipGraphicsResource *lightsMatrixResource, hipGraphicsResource *lightsScanSumResource)
{
	// Create modelview and projection matrix
	glm::mat4 m = glm::mat4(gl_ModelViewMatrix[0],gl_ModelViewMatrix[1],gl_ModelViewMatrix[2],gl_ModelViewMatrix[3],gl_ModelViewMatrix[4],gl_ModelViewMatrix[5],gl_ModelViewMatrix[6],gl_ModelViewMatrix[7],gl_ModelViewMatrix[8],gl_ModelViewMatrix[9],gl_ModelViewMatrix[10],gl_ModelViewMatrix[11],gl_ModelViewMatrix[12],gl_ModelViewMatrix[13],gl_ModelViewMatrix[14],gl_ModelViewMatrix[15]);
	glm::mat4 proj = glm::mat4(gl_ProjectionMatrix[0],gl_ProjectionMatrix[1],gl_ProjectionMatrix[2],gl_ProjectionMatrix[3],gl_ProjectionMatrix[4],gl_ProjectionMatrix[5],gl_ProjectionMatrix[6],gl_ProjectionMatrix[7],gl_ProjectionMatrix[8],gl_ProjectionMatrix[9],gl_ProjectionMatrix[10],gl_ProjectionMatrix[11],gl_ProjectionMatrix[12],gl_ProjectionMatrix[13],gl_ProjectionMatrix[14],gl_ProjectionMatrix[15]);

	// Update lights and initialize scan sum _d_lightScanSum, _d_lightsMatrix;
	hipGraphicsResourceGetMappedPointer((void**) &d_lightsScanSum, &size, lightsScanSumResource);
	hipGraphicsResourceGetMappedPointer((void**) &d_lightsMatrixCompact, &size, lightsMatrixResource);
	hipMemcpy(d_pla,pointLightsArr, nLights*12*sizeof(float),hipMemcpyHostToDevice);
	hipMemset(d_lightsScanSum, 0, lightsScanSumLength*sizeof(int));

	// Begin kernell calls
	int nBlocks = glm::ceil((float)nLights/BLOCK_SIZE);
	
	calcLightProj<<<nBlocks,BLOCK_SIZE>>>(d_pla, threshold, right, m, proj, d_lightsProj, w, h, nLights); 

	calcLightsMatrix<<<nBlocks,BLOCK_SIZE>>>(d_lightsProj, d_lightsMatrix, nLights, gridRes, lightsTile, gLightsRow, gLightsCol); 

	nBlocks = glm::ceil((float)lightsScanSumLength/BLOCK_SIZE);

	preScanSum<<<nBlocks,BLOCK_SIZE>>>(d_lightsScanSum, d_lightsMatrix, lightsScanSumLength, lightsTile, nLights);

	thrust::inclusive_scan(thrust::device, d_lightsScanSum, d_lightsScanSum + lightsScanSumLength, d_lightsScanSum);

	compactLightsMatrix<<<nBlocks,BLOCK_SIZE>>>(d_lightsScanSum, d_lightsMatrix, d_lightsMatrixCompact, lightsTile, lightsScanSumLength, nLights); 
}