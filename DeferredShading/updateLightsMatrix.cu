#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include "glm.hpp"

#define BLOCK_SIZE				256

int *d_lightsScanSum, *d_lightsMatrix, *d_lightsMatrixCompact;
float *d_pla;
glm::vec3 *d_lightsProj;
hipError_t err;

__device__ float calcLightRadius(float* l, float threshold)
{
	if (l[11] == 0.0f) return (threshold*l[3]*glm::max(glm::max(l[0],l[1]),l[2])-l[9])/l[10];
	else return (sqrt(-4*l[9]*l[11]+l[10]*l[10]+4*l[11]*threshold*l[3]*glm::max(glm::max(l[0],l[1]),l[2]))-l[10])/(2*l[11]);
}


__global__ void calcLightProj(float *d_pla, float threshold, glm::vec3 right, glm::mat4 m, glm::mat4 proj, glm::vec3* d_lightsProj, int w, int h, int nLights)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;

	if (idx < nLights) {
		glm::vec4 c = glm::vec4(d_pla[idx*12+6], d_pla[idx*12+7], d_pla[idx*12+8],1.0);
		float r = calcLightRadius(&d_pla[idx*12], threshold);
	
		glm::vec4 p = c+r*glm::vec4(right,0.0);
		p.w=1.0f;

		glm::vec4 cp = proj*m*c;
		cp = cp/cp.w;
		cp.x = w*(cp.x+1)/2;
		cp.y = h*(cp.y+1)/2;
	
		glm::vec4 pp = proj*m*p;
		pp = pp/pp.w;
		pp.x = w*(pp.x+1)/2;
		pp.y = h*(pp.y+1)/2;
	
		float pRadius = glm::length(glm::vec2(cp.x,cp.y)-glm::vec2(pp.x,pp.y));

		d_lightsProj[idx] = glm::vec3(cp.x,cp.y,pRadius);
	}
}

__global__ void calcLightsMatrix(glm::vec3* lightsProj, int* lightsScanSum, int* lightsMatrix, int nLights, int gridRes, int lightsTile, int gLightsCol)
{
	int j = blockIdx.x;
	int k = blockIdx.y;
	int x1 = k*gridRes; int x2 = (k+1)*gridRes;
	int y1 = j*gridRes; int y2 = (j+1)*gridRes;
	int off;
	for (int i = 0; i < nLights ; ++i)
	{
		glm::vec2 cp = glm::vec2(lightsProj[i].x,lightsProj[i].y);
		float pRadius = lightsProj[i].z;
		if (cp.x >= x1 && cp.x < x2) {
			if (cp.y >= y1 && cp.y < y2) { //center inside tile
				off = lightsScanSum[j*gLightsCol+k]++; // comprovacio que no ens pasem de max num ligths per tile
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			} 
			else if (cp.y > y2) { // down (r> y-y2)
				if (pRadius >= abs(cp.y-y2)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			} 
			else if (cp.y < y1) { // up (r> y-y1)
				if (pRadius >= abs(cp.y-y1)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			} 
		} 
		else if (cp.y >= y1 && cp.y < y2) { 
			if (cp.x < x1) { // left
				if (pRadius >= abs(cp.x-x1)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			}
			else if (cp.x > x2) { // right
				if (pRadius >= abs(cp.x-x2)) {
					off = lightsScanSum[j*gLightsCol+k]++;
					lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
				}
			}
		}
		else if (cp.x < x1 && cp.y < y1) { // upper-left
			if (pRadius >= sqrt((cp.x-x1)*(cp.x-x1)+(cp.y-y1)*(cp.y-y1))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		} 
		else if (cp.x > x2 && cp.y < y1) { // upper-right
			if (pRadius >= sqrt((cp.x-x2)*(cp.x-x2)+(cp.y-y1)*(cp.y-y1))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		}
		else if (cp.x < x1 && cp.y > y2) { // down-left
			if (pRadius >= sqrt((cp.x-x1)*(cp.x-x1)+(cp.y-y2)*(cp.y-y2))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		}
		else if (cp.x > x2 && cp.y > y2) { // down-right
			if (pRadius >= sqrt((cp.x-x2)*(cp.x-x2)+(cp.y-y2)*(cp.y-y2))) {
				off = lightsScanSum[j*gLightsCol+k]++;
				lightsMatrix[(j*gLightsCol+k)*lightsTile+off] = i;
			}
		}
	}
}

__global__ void compactLightsMatrix(int* lightsScanSum, int* lightsMatrix, int* lightsMatrixCompact, int lightsTile, int lightsScanSumLength)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < lightsScanSumLength) {
		int prev;
		if (idx == 0) prev = 0;
		else prev = lightsScanSum[idx-1];
		int act = lightsScanSum[idx];
		for (int i = 0; i < act-prev; ++i) {
			lightsMatrixCompact[prev+i] = lightsMatrix[idx*lightsTile+i];
		}
	}
}

extern "C" void initMemCUDA(int nLights, int lightsScanSumLength, int lightsMatrixLength)
{
	err=hipMalloc((void **) &d_pla, nLights*12*sizeof(float)); //[r,g,b, i,i_,i__, x,y,z, con,lin,exp]
	err=hipMalloc((void **) &d_lightsProj, nLights*sizeof(glm::vec3)); // [x,y,radius]
	err=hipMalloc((void **) &d_lightsScanSum, lightsScanSumLength*sizeof(int));
	err=hipMalloc((void **) &d_lightsMatrix, lightsMatrixLength*sizeof(int));
	err=hipMalloc((void **) &d_lightsMatrixCompact, lightsMatrixLength*sizeof(int));
}

extern "C" void freeMemCUDA()
{
	hipFree(d_pla);;
	hipFree(d_lightsProj);
	hipFree(d_lightsMatrixCompact);
	hipFree(d_lightsScanSum);
	hipFree(d_lightsMatrix);
}

extern "C" void launch_kernel(void* pointLightsArr, int nLights, float threshold, glm::vec3 right, float* gl_ModelViewMatrix, float* gl_ProjectionMatrix, int w, int h,
	int gLightsRow, int gLightsCol, int* lightsScanSum, int lightsScanSumLength, int* lightsMatrix, int lightsMatrixLength, int gridRes, int lightsTile)
{
	// Create modelview and projection matrix
	glm::mat4 m = glm::mat4(gl_ModelViewMatrix[0],gl_ModelViewMatrix[1],gl_ModelViewMatrix[2],gl_ModelViewMatrix[3],gl_ModelViewMatrix[4],gl_ModelViewMatrix[5],gl_ModelViewMatrix[6],gl_ModelViewMatrix[7],gl_ModelViewMatrix[8],gl_ModelViewMatrix[9],gl_ModelViewMatrix[10],gl_ModelViewMatrix[11],gl_ModelViewMatrix[12],gl_ModelViewMatrix[13],gl_ModelViewMatrix[14],gl_ModelViewMatrix[15]);
	glm::mat4 proj = glm::mat4(gl_ProjectionMatrix[0],gl_ProjectionMatrix[1],gl_ProjectionMatrix[2],gl_ProjectionMatrix[3],gl_ProjectionMatrix[4],gl_ProjectionMatrix[5],gl_ProjectionMatrix[6],gl_ProjectionMatrix[7],gl_ProjectionMatrix[8],gl_ProjectionMatrix[9],gl_ProjectionMatrix[10],gl_ProjectionMatrix[11],gl_ProjectionMatrix[12],gl_ProjectionMatrix[13],gl_ProjectionMatrix[14],gl_ProjectionMatrix[15]);

	// Update lights and initialize scan sum
	err=hipMemcpy(d_pla,pointLightsArr, nLights*12*sizeof(float),hipMemcpyHostToDevice);
	err=hipMemset(d_lightsScanSum, 0, lightsScanSumLength*sizeof(int));

	// Begin kernell calls
	int nBlocks = glm::ceil((float)nLights/BLOCK_SIZE);
	
	calcLightProj<<<nBlocks,BLOCK_SIZE>>>(d_pla, threshold, right, m, proj, d_lightsProj, w, h, nLights); //O(1)

	calcLightsMatrix<<<dim3(gLightsRow, gLightsCol),1>>>(d_lightsProj, d_lightsScanSum, d_lightsMatrix, nLights, gridRes, lightsTile, gLightsCol); //O(n)

	thrust::inclusive_scan(thrust::device, d_lightsScanSum, d_lightsScanSum + lightsScanSumLength, d_lightsScanSum);

	nBlocks = glm::ceil((float)lightsScanSumLength/BLOCK_SIZE);

	compactLightsMatrix<<<nBlocks,BLOCK_SIZE>>>(d_lightsScanSum, d_lightsMatrix, d_lightsMatrixCompact, lightsTile, lightsScanSumLength); //O(1)

	// Copy results
	hipMemcpy(lightsScanSum ,d_lightsScanSum, lightsScanSumLength*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(lightsMatrix ,d_lightsMatrixCompact, lightsMatrixLength*sizeof(int),hipMemcpyDeviceToHost);
}